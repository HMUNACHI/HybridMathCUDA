#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

/*

CUDA kernels that calculates the following in paralle on GPU for a batch of inputs:

- The derivative of sin - sin'(x) = (sin(x + h) - sin(x - h)) / 2h
- The integral of sin - ∫sin(x)dx = h * (sin(x) + sin(x - h)) / 2
- The linearization of sin - sin(x) ≈ sin(x0) + sin'(x0) * (x - x0)
- Finding the zero of sin Newton-Raphson method for sin - x = x - (sin(x) - x) / (cos(x) - 1)

The zero of a function is any value within the function's domain that, when input into the function, 
results in an output of zero. In other words, it's the x-value (or input value) 
where the function's graph intersects the x-axis.

N/B: The Newton-Raphson only calculates the first 10 iteration results for demonstrative purposes.
When we take the Newton-Raphson of the derivative of the function, we are finding the zero of the derivative.
That is the point where the derivative (chhange of the function is at a maximum or minimum), aka optimisation.

This is the bedrock of optimization. You can run on even higher order derivatives.

If you use the calculate derivative kernel to calculate the derivative of a function, you can use again  to 
calculate the derivative of its outputs and so on.
*/

__global__ void calculateDerivative(float* input, float* output, float h, int numElements) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx > 0 && idx < numElements - 1) {
        output[idx] = (input[idx + 1] - input[idx - 1]) / (2 * h);
    }
}

__global__ void calculateIntegral(float* input, float* output, float h, int numElements) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx > 0 && idx < numElements - 1) {
        output[idx] = h * (input[idx - 1] + input[idx]) / 2;
    }
}

__global__ void calculateLinearization(float* input, float* derivative, float* output, float h, int numElements) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx > 0 && idx < numElements - 1) {
        float x0 = idx * h;
        output[idx] = input[idx] + derivative[idx] * (x0 - idx * h);
    }
}

__global__ void calculateNewtonRaphson(float* input, float* output, float h, int numElements) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx > 0 && idx < numElements - 1) {
        float x0 = input[idx];
        for (int i = 0; i < 10; i++) { // Perform 10 iterations
            x0 = x0 - (sinf(x0) - x0) / (cosf(x0) - 1);
        }
        output[idx] = x0;
    }
}

int main() {
    int numElements = 1000;
    float h = 0.01f;
    size_t size = numElements * sizeof(float);

    float* hostInput = (float*)malloc(size);
    float* hostOutputDerivative = (float*)malloc(size);
    float* hostOutputIntegral = (float*)malloc(size);
    float* hostOutputLinearization = (float*)malloc(size);
    float* hostOutputNewtonRaphson = (float*)malloc(size);

    for (int i = 0; i < numElements; i++) {
        hostInput[i] = sinf(i * h);
    }

    float* deviceInput;
    float* deviceOutputDerivative;
    float* deviceOutputIntegral;
    float* deviceOutputLinearization;
    float* deviceOutputNewtonRaphson;

    hipMalloc((void**)&deviceInput, size);
    hipMalloc((void**)&deviceOutputDerivative, size);
    hipMalloc((void**)&deviceOutputIntegral, size);
    hipMalloc((void**)&deviceOutputLinearization, size);
    hipMalloc((void**)&deviceOutputNewtonRaphson, size);

    hipMemcpy(deviceInput, hostInput, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    calculateDerivative<<<blocksPerGrid, threadsPerBlock>>>(deviceInput, deviceOutputDerivative, h, numElements);
    calculateIntegral<<<blocksPerGrid, threadsPerBlock>>>(deviceInput, deviceOutputIntegral, h, numElements);
    calculateLinearization<<<blocksPerGrid, threadsPerBlock>>>(deviceInput, deviceOutputDerivative, deviceOutputLinearization, h, numElements);
    calculateNewtonRaphson<<<blocksPerGrid, threadsPerBlock>>>(deviceInput, deviceOutputNewtonRaphson, h, numElements);

    hipMemcpy(hostOutputDerivative, deviceOutputDerivative, size, hipMemcpyDeviceToHost);
    hipMemcpy(hostOutputIntegral, deviceOutputIntegral, size, hipMemcpyDeviceToHost);
    hipMemcpy(hostOutputLinearization, deviceOutputLinearization, size, hipMemcpyDeviceToHost);
    hipMemcpy(hostOutputNewtonRaphson, deviceOutputNewtonRaphson, size, hipMemcpyDeviceToHost);

    // Print some values for verification
    for (int i = 0; i < 10; i++) {
        std::cout << "Sine Output: " << hostInput[i] << std::endl;
        std::cout << "Derivative: " << hostOutputDerivative[i] << std::endl;
        std::cout << "Integral: " << hostOutputIntegral[i] << std::endl;
        std::cout << "Linearization: " << hostOutputLinearization[i] << std::endl;
        std::cout << "Newton-Raphson: " << hostOutputNewtonRaphson[i] << "\n" << std::endl;

    }

    free(hostInput);
    free(hostOutputDerivative);
    free(hostOutputIntegral);
    free(hostOutputLinearization);

    hipFree(deviceInput);
    hipFree(deviceOutputDerivative);
    hipFree(deviceOutputIntegral);
    hipFree(deviceOutputLinearization);

    return 0;
}