#include <iostream> // for std::cout
#include <hip/hip_runtime.h> // for CUDA runtime functions
 // for CUDA kernel launch parameters

/* 

Addition of Two Vectors: 
--------------------------
a + b = (a1 + b1, a2 + b2, ..., an + bn)

*/
__global__ void vectorAdd(const int *a, 
                          const int *b, 
                          int *c, 
                          int numElements) {
    // Calculate the global index for the current thread
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure we don't go out of bounds
    if (i < numElements){
        // Add corresponding elements of input vectors
        c[i] = a[i] + b[i];
    }
}

int main() {
    // Number of elements in vectors
    int numElements = 50;

    // Calculate the size of the data
    size_t size = numElements * sizeof(int);

    // Declare pointers for host and device vectors
    int *hostA, *hostB, *hostC;
    int *deviceA, *deviceB, *deviceC;

    // Allocate memory for host vectors
    hostA = (int*) malloc(size);
    hostB = (int*) malloc(size);
    hostC = (int*) malloc(size);

    // Initialize input vectors
    for (int i = 0; i < numElements; i++){
        hostA[i] = i;
        hostB[i] = i*2;
    }

    // Allocate memory for device vectors
    hipMalloc((void**)&deviceA, size);
    hipMalloc((void**)&deviceB, size);
    hipMalloc((void**)&deviceC, size);

    // Copy host vectors to device
    hipMemcpy(deviceA, hostA, size, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the vectorAdd kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(deviceA, deviceB, deviceC, numElements);

    // Copy result vector from device to host
    hipMemcpy(hostC, deviceC, size, hipMemcpyDeviceToHost);

    // Print the results
    for (int i = 0; i < numElements; i++){
        std::cout << hostA[i] << " + " << hostB[i] << " = " << hostC[i] << std::endl;
    }

    // Free the memory allocated for host vectors
    free(hostA);
    free(hostB);
    free(hostC);

    // Free the memory allocated for device vectors
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return 0;
}