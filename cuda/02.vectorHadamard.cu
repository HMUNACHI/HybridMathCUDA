// Include necessary libraries
#include <ctime>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>


// CUDA kernel for element-wise multiplication of two vectors aka Hadamard product
__global__ void vectorMul(const float *a,
                          const float *b,
                          float *c,
                          float numElements){
    // Calculate the unique index for the thread
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // Check if index is within the range of elements
    if (idx < numElements){
        // Perform the multiplication
        c[idx] = a[idx] * b[idx];
    }
}

// Function to generate a random float
float randomFloat(int randMax = 1000){
    // Return a random float between 0 and 1
    return static_cast<float>(rand()) / static_cast<float>(randMax);
}

// Main function
int main(){
    // Seed the random number generator
    srand(time(0));

    // Define the number of elements in the vectors
    int numElements = 100;

    // Calculate the size of the vectors in bytes
    size_t size = numElements * sizeof(float);
    
    // Declare pointers for host and device vectors
    float *hostA, *hostB, *hostC;
    float *deviceA, *deviceB, *deviceC;

    // Allocate memory for host vectors
    hostA = (float*) malloc(size);
    hostB = (float*) malloc(size);
    hostC = (float*) malloc(size);

    // Initialize host vectors with random floats
    for (int idx = 0; idx < numElements; idx++){
        hostA[idx] = randomFloat();
        hostB[idx] = randomFloat();
    }

    // Allocate memory for device vectors
    hipMalloc((void**)&deviceA, size);
    hipMalloc((void**)&deviceB, size);
    hipMalloc((void**)&deviceC, size);

    // Copy host vectors to device
    hipMemcpy(deviceA, hostA, size, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, size, hipMemcpyHostToDevice);

    // Define the number of threads per block and the number of blocks per grid
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the vector multiplication kernel
    vectorMul<<<blocksPerGrid, threadsPerBlock>>>(deviceA, deviceB, deviceC, numElements);

    // Copy the result vector from device to host
    hipMemcpy(hostC, deviceC, size, hipMemcpyDeviceToHost);

    // Print the results of the multiplication
    for (int idx = 0; idx < numElements; idx++){
        std::cout << hostA[idx] << " * " << hostB[idx] << " = " << hostC[idx] << std::endl;
    }

    // Free the memory allocated for host vectors
    free(hostA);
    free(hostB);
    free(hostC);

    // Free the memory allocated for device vectors
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
}
