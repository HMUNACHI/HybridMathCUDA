// Include necessary libraries
#include <ctime>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>


/* 

CUDA kernels that calculates:
- The mean of a vector
- The standard deviation of a vector
- The normalisation of the vector - (xi - μ) / σ
- The skewness of the vector - (1/n) * Σ[(xi - μ) / σ]^3
- The kurtosis of the vector - (1/n) * Σ[(xi - μ) / σ]^4
- Variance (s²) = (1 / (n - 1)) * Σ(xi - x̄)²
- Covariance = (1 / (n - 1)) * Σ(xi - x̄)(yi - ȳ)

*/

__global__ void vectorMean(const float *vector, float *mean, int numElements){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < numElements){
        atomicAdd(mean, vector[idx]);
    }
}

__global__ void vectorStd(const float *vector, float *std, float *mean, int numElements){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < numElements){
        atomicAdd(std, pow(vector[idx] - *mean, 2));
    }
}

__global__ void vectorNormalization(float *vector, float mean, float std, int numElements){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < numElements){
        vector[idx] = (vector[idx] - mean) / std;
    }
}

__global__ void vectorSkewnessKurtosis(const float *vector, float *skewness, float *kurtosis, float mean, float std, int numElements){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < numElements){
        float standardized = (vector[idx] - mean) / std;
        atomicAdd(skewness, pow(standardized, 3));
        atomicAdd(kurtosis, pow(standardized, 4));
    }
}

__global__ void vectorVariance(const float *vector, float *variance, float mean, int numElements){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < numElements){
        atomicAdd(variance, pow(vector[idx] - mean, 2));
    }
}

__global__ void vectorCovariance(const float *vectorX, const float *vectorY, float *covariance, float meanX, float meanY, int numElements){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < numElements){
        atomicAdd(covariance, (vectorX[idx] - meanX) * (vectorY[idx] - meanY));
    }
}

float randomFloat(int randMax = 1000){
    return static_cast<float>(rand()) / static_cast<float>(randMax);
}

int main(){
    srand(time(0));
    int numElements = 300;
    size_t size = numElements * sizeof(float);
    float *hostVector, *deviceVector;
    float *hostNormalized, *deviceNormalized;
    float *hostMean, *deviceMean;
    float *hostStd, *deviceStd;
    float *hostSkewness, *deviceSkewness;
    float *hostKurtosis, *deviceKurtosis;
    float *hostVariance, *deviceVariance;
    float *hostCovariance, *deviceCovariance;

    hostVector = (float*) malloc(size);
    hostNormalized = (float*) malloc(size);
    hostMean = (float*) malloc(sizeof(float));
    hostStd = (float*) malloc(sizeof(float));
    hostSkewness = (float*) malloc(sizeof(float));
    hostKurtosis = (float*) malloc(sizeof(float));
    hostVariance = (float*) malloc(sizeof(float));
    hostCovariance = (float*) malloc(sizeof(float));

    // Initialise a random vector of distribution
    for (int idx = 0; idx < numElements; idx++){
        hostVector[idx] = randomFloat();
    }

    // Allocate device vector and variables
    hipMalloc((void**)&deviceVector, size);
    hipMalloc((void**)&deviceNormalized, size);
    hipMalloc((void**)&deviceMean, sizeof(float));
    hipMalloc((void**)&deviceStd, sizeof(float));
    hipMalloc((void**)&deviceSkewness, sizeof(float));
    hipMalloc((void**)&deviceKurtosis, sizeof(float));
    hipMalloc((void**)&deviceVariance, sizeof(float));
    hipMalloc((void**)&deviceCovariance, sizeof(float));

    // Copy host vectors to device
    hipMemcpy(deviceVector, hostVector, size, hipMemcpyHostToDevice);

    // Define the number of threads per block and the number of blocks per grid
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the vector mean kernel
    vectorMean<<<blocksPerGrid, threadsPerBlock>>>(deviceVector, deviceMean, numElements);
    hipDeviceSynchronize();
    hipMemcpy(hostMean, deviceMean, sizeof(float), hipMemcpyDeviceToHost);
    *hostMean /= numElements;

    // Launch the vector std kernel
    vectorStd<<<blocksPerGrid, threadsPerBlock>>>(deviceVector, deviceStd, deviceMean, numElements);
    hipDeviceSynchronize();
    hipMemcpy(hostStd, deviceStd, sizeof(float), hipMemcpyDeviceToHost);
    *hostStd = sqrt(*hostStd / numElements);

    // Launch the vector normalization kernel
    vectorNormalization<<<blocksPerGrid, threadsPerBlock>>>(deviceVector, *hostMean, *hostStd, numElements);
    hipDeviceSynchronize();
    hipMemcpy(hostNormalized, deviceVector, size, hipMemcpyDeviceToHost);

    // Launch the vector skewness and kurtosis kernel
    vectorSkewnessKurtosis<<<blocksPerGrid, threadsPerBlock>>>(deviceVector, deviceSkewness, deviceKurtosis, *hostMean, *hostStd, numElements);
    hipDeviceSynchronize();
    hipMemcpy(hostSkewness, deviceSkewness, sizeof(float), hipMemcpyDeviceToHost);

    // Launch the vector variance kernel
    vectorVariance<<<blocksPerGrid, threadsPerBlock>>>(deviceVector, deviceVariance, *hostMean, numElements);
    hipDeviceSynchronize();
    hipMemcpy(hostVariance, deviceVariance, sizeof(float), hipMemcpyDeviceToHost);

    // Launch the vector covariance kernel
    vectorCovariance<<<blocksPerGrid, threadsPerBlock>>>(deviceVector, deviceNormalized, deviceCovariance, *hostMean, *hostMean, numElements);
    hipDeviceSynchronize();
    hipMemcpy(hostCovariance, deviceCovariance, sizeof(float), hipMemcpyDeviceToHost);

    // Print the results
    std::cout << "Mean: " << *hostMean << std::endl;
    std::cout << "Standard Deviation: " << *hostStd << std::endl;
    std::cout << "Skewness: " << *hostSkewness << std::endl;
    std::cout << "Kurtosis: " << *hostKurtosis << std::endl;
    std::cout << "Variance: " << *hostVariance << std::endl;
    std::cout << "Covariance: " << *hostCovariance << std::endl;

    // Free the memory allocated for host vectors
    free(hostVector);
    free(hostNormalized);
    free(hostMean);
    free(hostStd);
    free(hostSkewness);
    free(hostKurtosis);
    free(hostVariance);
    free(hostCovariance);

    // Free the memory allocated for device vectors
    hipFree(deviceVector);
    hipFree(deviceNormalized);
    hipFree(deviceMean);
    hipFree(deviceStd);
    hipFree(deviceSkewness);
    hipFree(deviceKurtosis);
    hipFree(deviceVariance);
    hipFree(deviceCovariance);

}
