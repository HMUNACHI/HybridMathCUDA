// Include necessary libraries
#include <ctime>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>


/*

Correlation Coefficient of Two Vectors:
----------------------------------------    
Vector correlation is a measure of the relationship between two or more variables.
Where -1 indicates a perfect negative correlation, 0 indicates no correlation, and 1 indicates a perfect positive correlation.
The formula for the correlation coefficient is given by: 

Pearson r = Σ((xi - x̄)(yi - ȳ)) / √(Σ(xi - x̄)² * Σ(yi - ȳ)²)

Cosine similarity focuses on the angle between vectors, measuring how similar their directions are regardless of magnitude.
While correlation takes into account the mean and standard deviation of the variables; how the variables change together.
Not their angle of inclination when plotted in a multi-dimensional space.

This CUDA demonstrates how to breakdown complex equations into multiple sub-kernels and yet parallelise.
However, launching kernels takes overhead cost, so it's not always beneficial to break down into multiple kernels.

*/

__global__ void calculateNumerator(const float *a, const float *b, float *numerator, float meanA, float meanB, int numElements) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < numElements){
        atomicAdd(numerator, (a[idx] - meanA) * (b[idx] - meanB));
    }
}

__global__ void calculateDenominator(const float *vector, float *denominator, float meanB, int numElements) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < numElements){
        atomicAdd(denominator, (vector[idx] - meanB) * (vector[idx] - meanB));
    }
}

float randomFloat(int randMax = 1000){
    return static_cast<float>(rand()) / static_cast<float>(randMax);
}

float calculateMean(float* vector, int numElements){
    float sum = 0;
    for(int i = 0; i < numElements; i++){
        sum += vector[i];
    }
    return sum / numElements;
}

int main(){
    srand(time(0));
    int numElements = 300;
    size_t size = numElements * sizeof(float);
    float *hostVectorA, *hostVectorB, *hostNumerator, *hostDenominatorA, *hostDenominatorB;
    float *deviceVectorA, *deviceVectorB, *deviceNumerator, *deviceDenominatorA, *deviceDenominatorB;

    hostVectorA = (float*) malloc(size);
    hostVectorB = (float*) malloc(size);
    hostNumerator = (float*) malloc(sizeof(float));
    hostDenominatorA = (float*) malloc(sizeof(float));
    hostDenominatorB = (float*) malloc(sizeof(float));

    for (int idx = 0; idx < numElements; idx++){
        hostVectorA[idx] = randomFloat();
        hostVectorB[idx] = randomFloat();
    }

    float meanA = calculateMean(hostVectorA, numElements);
    float meanB = calculateMean(hostVectorB, numElements);

    hipMalloc((void**)&deviceVectorA, size);
    hipMalloc((void**)&deviceVectorB, size);
    hipMalloc((void**)&deviceNumerator, sizeof(float));
    hipMalloc((void**)&deviceDenominatorA, sizeof(float));
    hipMalloc((void**)&deviceDenominatorB, sizeof(float));

    hipMemcpy(deviceVectorA, hostVectorA, size, hipMemcpyHostToDevice);
    hipMemcpy(deviceVectorB, hostVectorB, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    calculateNumerator<<<blocksPerGrid, threadsPerBlock>>>(deviceVectorA, deviceVectorB, deviceNumerator, meanA, meanB, numElements);
    calculateDenominator<<<blocksPerGrid, threadsPerBlock>>>(deviceVectorA, deviceDenominatorA, meanA, numElements);
    calculateDenominator<<<blocksPerGrid, threadsPerBlock>>>(deviceVectorB, deviceDenominatorB, meanB, numElements);

    hipMemcpy(hostNumerator, deviceNumerator, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostDenominatorA, deviceDenominatorA, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostDenominatorB, deviceDenominatorB, sizeof(float), hipMemcpyDeviceToHost);

    float correlationCoefficient = *hostNumerator / sqrt(*hostDenominatorA * *hostDenominatorB);

    std::cout << "Correlation Coefficient: " << correlationCoefficient << std::endl;

    free(hostVectorA);
    free(hostVectorB);
    free(hostNumerator);
    free(hostDenominatorA);
    free(hostDenominatorB);

    hipFree(deviceVectorA);
    hipFree(deviceVectorB);
    hipFree(deviceNumerator);
    hipFree(deviceDenominatorA);
    hipFree(deviceDenominatorB);
}